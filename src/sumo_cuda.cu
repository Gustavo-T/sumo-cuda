#include "hip/hip_runtime.h"
/**
 * @file: sumo_cuda.cu
 * @author: Chris Blatchley
 * @author: Thad Bond
 *
 * Main entry point for sumo-cuda
 */
#include <cstdio>
#include "network.cuh"
#include "junction.cuh"
#include "edge.cuh"
#include "route.cuh"
#include "vehicle_control.cuh"

void printHelpString()
{
    printf("SUMO-CUDA\n");
    printf("    USAGE: sumo-cuda [options] network.netccfg\n");
    printf("\n");
    printf("Authors: Thaddeus Bond, Chris Blatchley\n");
}

void test()
{
    Network network = Network(150);
    Junction * j1 = network.addJunction( Junction::AllStop );
    Edge *e1 = network.addEdge( 1000.00, 30.0, j1 );
    Route *r1 = network.addRoute();
    r1->addEdge(e1);
    Vehicle::Style style = {5.0, 30.0};
    (network.vehicleController)->queueVehicle(r1, style, 5);
    network.runSimulation();
}

int main(int argc, char const *argv[])
{
    test();
    return 0;
}

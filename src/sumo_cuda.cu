#include "hip/hip_runtime.h"
/**
 * @file: sumo_cuda.cu
 * @author: Chris Blatchley
 * @author: Thad Bond
 *
 * Main entry point for sumo-cuda
 */
#include <cstdio>
#include <hash_map>
#include <string>
#include <stdlib.h>
#include <sstream>
#include "network.cuh"
#include "junction.cuh"
#include "edge.cuh"
#include "route.cuh"
#include "vehicle_control.cuh"
#include "tinyxml2.h"

//Helper functions for splitting a delimited string
//Source: http://stackoverflow.com/a/236803
std::vector<std::string> &split(const std::string &s, char delim, std::vector<std::string> &elems) {
    std::stringstream ss(s);
    std::string item;
    while (std::getline(ss, item, delim)) {
        elems.push_back(item);
    }
    return elems;
}


std::vector<std::string> split(const std::string &s, char delim) {
    std::vector<std::string> elems;
    split(s, delim, elems);
    return elems;
}

void printHelpString()
{
    printf("SUMO-CUDA\n");
    printf("    USAGE: sumo-cuda [options] network.netccfg\n");
    printf("\n");
    printf("Authors: Thaddeus Bond, Chris Blatchley\n");
}

void test()
{
    Network network = Network(0,150);
    Junction * j1 = network.addJunction( Junction::AllStop );
    Edge *e1 = network.addEdge( 1000.00, 30.0, j1 );
    Route *r1 = network.addRoute();
    r1->addEdge(e1);
    Vehicle::Style style = {5.0, 30.0};
    (network.vehicleController)->queueVehicle(r1, style, 5);
    network.runSimulation();
}

void runFile(const char * cfgFile)
{
	tinyxml2::XMLDocument cfgDoc;
	cfgDoc.LoadFile(cfgFile);

	//Get the cfg node
	tinyxml2::XMLNode * cfgNode = cfgDoc.FirstChildElement();

	const char * netFile = cfgNode->FirstChildElement("input")->FirstChildElement("net-file")->Attribute("value");
	const char * routeFile = cfgNode->FirstChildElement("input")->FirstChildElement("route-files")->Attribute("value");
	int startTime = strtol(cfgNode->FirstChildElement("time")->FirstChildElement("begin")->Attribute("value"), NULL, 10);
	int endTime = strtol(cfgNode->FirstChildElement("time")->FirstChildElement("end")->Attribute("value"), NULL, 10);
	//Define the network object. "You are an amazing object, Network, good to have you!"
	Network network = Network(startTime, endTime);

	tinyxml2::XMLDocument netDoc;
	netDoc.LoadFile(netFile);

	//Get the "net" node
	tinyxml2::XMLNode * netNode = netDoc.FirstChildElement();

	//Create a junction map so we can set up the network via the proper ids
	std::hash_map<std::string, Junction*> junctionMap;
	//Loop through and create all the junction objects
	for(tinyxml2::XMLElement * j = netNode->FirstChildElement("junction"); j != NULL; j = j->NextSiblingElement("junction"))
	{
		//Default shape Throughway
		Junction::Shape shape = Junction::Throughway;
		if(j->Attribute("type") == "allway_stop")
		{
			shape = Junction::AllStop;
		}else if(j->Attribute("type") == "unregulated")
		{
			shape = Junction::Throughway;
		}

		//Create junction and store it
		Junction * junction = network.addJunction( shape );
		junctionMap.insert( std::pair<std::string, Junction*>(j->Attribute("id"), junction) );
	}

	//Create our edge map so we can find edges by id for setup
	std::hash_map<std::string, Edge*> edgeMap;
	//Loop through all the edge nodes
	for(tinyxml2::XMLElement * e = netNode->FirstChildElement("edge"); e != NULL; e = e->NextSiblingElement("edge"))
	{
		//Create the edge node with the appropriate values and junction ending
		if(e->Attribute("to") == NULL)
		{
			//This is an internal edge, skip it.
			continue;
		}
		Edge * edge = network.addEdge(e->FirstChildElement()->FloatAttribute("length"), e->FirstChildElement()->FloatAttribute("speed"), junctionMap[e->Attribute("to")]);

		for(tinyxml2::XMLElement * l = e->FirstChildElement("lane"); l != NULL; l = l->NextSiblingElement("lane"))
		{
			if(l != e->FirstChildElement("lane"))
			{
				//If we aren't the first lane, add another
				edge->addLane();
			}
		}

		//Insert the edge into our map for further use
		edgeMap.insert( std::pair<std::string, Edge*>(e->Attribute("id"), edge) );
	}

	tinyxml2::XMLDocument routeDoc;
	routeDoc.LoadFile(routeFile);

	//Get the "net" node
	tinyxml2::XMLNode * routeNode = routeDoc.FirstChildElement();

	//Create a map so we can find routes by ids for setup
	std::hash_map<std::string, Route*> routeMap;
	//First, create all our routes within the network
	for(tinyxml2::XMLElement * r = routeNode->FirstChildElement("route"); r != NULL; r = r->NextSiblingElement("route"))
	{
		//Create a route object
		Route * route = new Route();
		std::vector<std::string> edgeList = split( r->Attribute("edges"), ' ' );
		for(std::vector<std::string>::iterator it = edgeList.begin(); it != edgeList.end(); ++it) {
			//Add that edge to the route
			route->addEdge(edgeMap[*it]);
		}

		routeMap.insert( std::pair<std::string, Route*>(r->Attribute("id"), route) );
	}

	//Create a map of vehicle styles to be used in vehicle queueing
	std::hash_map<std::string, Vehicle::Style> styleMap;
	//Loop through all the vehicle styles
	for(tinyxml2::XMLElement * s = routeNode->FirstChildElement("vType"); s != NULL; s = s->NextSiblingElement("vType"))
	{
		Vehicle::Style style = {std::strtod(s->Attribute("length"), NULL), std::strtod(s->Attribute("maxSpeed"), NULL)};
		styleMap.insert( std::pair<std::string, Vehicle::Style>(s->Attribute("id"), style) );
	}

	for(tinyxml2::XMLElement * v = routeNode->FirstChildElement("vehicle"); v != NULL; v = v->NextSiblingElement("vehicle"))
	{
		(network.vehicleController)->queueVehicle(routeMap[v->Attribute("route")], styleMap[v->Attribute("type")], v->IntAttribute("depart"));
	}

	network.runSimulation();
	
}

int main(int argc, char const *argv[])
{
	if(argc == 2)
	{
		runFile(argv[1]);
	}else{
		test();
	}
    return 0;
}



